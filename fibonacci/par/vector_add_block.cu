#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>


#define N 64
#define MAX_ERR 1e-6

__global__ void vector_add(float *out, float *a, float *b, int power) {
  //int stride = 1;	
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
           //  0        *    256     +    1 = 1  | BLOCK0 |  
           //  0        *    256     +    2 = 2 

           //  1        *    256     +    1 = 257 | BLOCK1 |   
           //  1        *    256     +    2 = 258

  //out[tid] = a[tid] + b[tid];  
  float golden = 1.61803398875;	
  
  float golden_to_power = pow(golden,power);
  float golden_minus_one_to_power = pow((1 - golden),power); 

  out[tid] = golden_to_power - golden_minus_one_to_power;
}

int main(){
    float *a, *b, *out;
    float *d_a, *d_b, *d_out; 

    // Allocate host memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);


    // Allocate device memory
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    // Executing kernel 
    int power = 100;
    vector_add<<<1,1>>>(d_out, d_a, d_b, power);
    
    // Transfer data back to host memory
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    float result = out[0]/2.23606797749979;
    printf("out[0] = %lF\n", result);
    //printf("PASSED\n");

    // Deallocate device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    // Deallocate host memory
    free(a); 
    free(b); 
    free(out);
}
