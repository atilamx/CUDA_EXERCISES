#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


#define N 512
#define MAX_ERR 1e-6

// __global__ matrix_add(int *out; int *A, int *B){


// }

void test_pass_function_array(int A[], int B[]){


}

int main(){
    
    //Create new matrix 
    int A = [1,2,3];
    int B = [2,3,3]; 

    int *d_a, *d_b, *d_out;

    // // Allocate device memory
    // hipMalloc((void**)&d_a, sizeof(int) * 3);
    // hipMalloc((void**)&d_b, sizeof(int) * 3);
    // hipMalloc((void**)&d_out, sizeof(int) * 3);
 
    // // Executing kernel 

    // matrix_add<<1,1>>>(d_out, d_a, d_b, N);
     
    test_pass_function_array(A, B);

    printf("out[0] = %f\n", out[0]);
    printf("PASSED\n");

}
