#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define N 512
#define MAX_ERR 1e-6

//__global__ void vector_add(float *out, float *a, float *b, int n) {
//   int stride = 1;	
//   int tid = blockIdx.x * blockDim.x + threadIdx.x;
           //  0        *    256     +    1 = 1  | BLOCK0 |  
           //  0        *    256     +    2 = 2 

           //  1        *    256     +    1 = 257 | BLOCK1 |   
           //  1        *    256     +    2 = 258

//   out[tid] = a[tid] + b[tid];   
//}

void print_results(float *C){
  printf("[");
  for(int i = 0 ; i < 20; i++){
    printf("%f,",C[i]);
  }
  printf("]\n");
}

__global__ void vector_dot_product(float *CUDA_A, float *CUDA_B, float *CUDA_C,int n) {
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  __shared__ float SHARED_SUM[4];
  extern __shared__ float SHARED_PRODUCTS[4];
  int row = 3;
  int col = 3;
  float mul[6];
  float res;
  //Data is not visible to threads in other blocks you idiot!
  for(int i = 0; i < 3; i++){
    //we need to flatten the stupid array 
    //mul[i] = CUDA_A[bid * row + (bid * col) + i] * CUDA_B[ i * row + tid];
    mul[i] = CUDA_A[bid * col + i] * CUDA_B[ i * row + tid];

  }

  res = mul[0] + mul[1] + mul[2];
  
  // Only one kernel should apply the dot product
  __syncthreads();
 
  //CUDA_C[(bid*2) + tid] = local_address[tid];
  //if (bid == 0) { 
    CUDA_C[(bid*3) + tid] = res;
  //}
}

void print_vectors(){
  printf("A=[{2.0,5.0,2.0}\n");
  printf("   {1.0,2.0,4.0}]\n");
  printf("   {4.0,2.0,2.0}]\n");
  printf("B=[{3.0,5.0,3.0}\n");
  printf("   {2.0,3.0,5.0}]\n");
  printf("   {7.0,1.0,3.0}]\n");
};

int main(){
    float *C;
    float *CUDA_A, *CUDA_B,  *CUDA_C; 

    // Allocate host memory
    float A[3][3] = {
      {2.0,5.0,2.0},
      {1.0,2.0,4.0},
      {4.0,2.0,2.0}
    };

    float B[3][3] = {
      {3.0,5.0,3.0},
      {2.0,3.0,5.0},
      {7.0,1.0,3.0}
    };
    
    C = (float *)malloc(3*3*sizeof(float) * N);

    // Allocate device memory
    hipMalloc((void**)&CUDA_A, sizeof(float) * N);
    hipMalloc((void**)&CUDA_B, sizeof(float) * N);
    hipMalloc((void**)&CUDA_C, sizeof(float) * N);

    // Transfer data from host to device memory
    hipMemcpy(CUDA_A, A, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(CUDA_B, B, sizeof(float) * N, hipMemcpyHostToDevice);

    vector_dot_product<<<3,3>>>(CUDA_A, CUDA_B, CUDA_C, N);
    hipMemcpy(C, CUDA_C, sizeof(float) * 500, hipMemcpyDeviceToHost);
    puts("DOT_PRODUCT");
    print_vectors();
    print_results(C);


    //printf("\nDot product result %f\n", *K);
    // Deallocate device memory
    hipFree(CUDA_A);
    hipFree(CUDA_B);
    hipFree(CUDA_C);

    // Deallocate host memory
    //free(A); 
    //free(B); 
    free(C);
}

