#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "ruby.h"
#include "extconf.h"
#define N 512
#define MAX_ERR 1e-6

//__global__ void vector_add(float *out, float *a, float *b, int n) {
//   int stride = 1;	
//   int tid = blockIdx.x * blockDim.x + threadIdx.x;
           //  0        *    256     +    1 = 1  | BLOCK0 |  
           //  0        *    256     +    2 = 2 

           //  1        *    256     +    1 = 257 | BLOCK1 |   
           //  1        *    256     +    2 = 258

//   out[tid] = a[tid] + b[tid];   
//}

void print_results(float *C){
  printf("ADDING\n");
  printf("[");
  for(int i = 0 ; i < 3; i++){
    printf("%f,",C[i]);
  }
  printf("]\n");
}

void print_results_sub(float *C){
  printf("SUBSTRACTING\n");

   printf("[");
   for(int i = 0 ; i < 3; i++){
     printf("%f,",C[i]);
   }
   printf("]\n");
}


__global__ void vector_add(float *CUDA_A, float *CUDA_B, float *CUDA_C, int n) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  CUDA_C[tid] = CUDA_A[tid] + CUDA_B[tid];   
}

__global__ void vector_sub(float *CUDA_A, float *CUDA_B, float *CUDA_C, int n) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  CUDA_C[tid] = CUDA_A[tid] - CUDA_B[tid];
}


extern "C"
void some(){
    float *C, *K;
    float *CUDA_A, *CUDA_B, *CUDA_C, *CUDA_K; 

    // Allocate host memory
    float A[3]= {2.0,4.0,6.0};
    printf("A = {2.0,4.0,6.0}\n");
    float B[3]= {1.0,2.0,3.0};
    printf("B = {1.0,2.0,3.0}\n");

    C = (float*)malloc(sizeof(float) * N);
    K = (float*)malloc(sizeof(float));

    // Allocate device memory
    hipMalloc((void**)&CUDA_A, sizeof(float) * N);
    hipMalloc((void**)&CUDA_B, sizeof(float) * N);
    hipMalloc((void**)&CUDA_C, sizeof(float) * N);
    hipMalloc((void**)&CUDA_C, sizeof(float) * N);
    hipMalloc((void**)&CUDA_K, sizeof(float));

    // Transfer data from host to device memory
    hipMemcpy(CUDA_A, A, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(CUDA_B, B, sizeof(float) * N, hipMemcpyHostToDevice);
    char str[100];
    int i;
    // Executing kernel 
    //for(;;){
     // printf( "Enter a value :");
     // scanf("%d",&i);

      //printf( "\nYou entered: %d \n",i);

      vector_add<<<1,3>>>(CUDA_A, CUDA_B, CUDA_C, N);
      hipMemcpy(C, CUDA_C, sizeof(float) * N, hipMemcpyDeviceToHost);

      //Executing kernel 

      print_results(C);

      vector_sub<<<1,3>>>(CUDA_A, CUDA_B, CUDA_C, N);
      hipMemcpy(C, CUDA_C, sizeof(float) * N, hipMemcpyDeviceToHost);

      print_results_sub(C);
    
    //}
    hipFree(CUDA_A);
    hipFree(CUDA_B);
    hipFree(CUDA_C);

    // Deallocate host memory
    //free(A); 
    //free(B); 
    free(C);
}

VALUE rb_print_length(VALUE self, VALUE str) {
  if (RB_TYPE_P(str, T_STRING) == 1) {
    some();	  
    return rb_sprintf("String length: %d", RSTRING_LEN(str));
  }

  return Qnil;
}

__global__ void simple_kernel(float *CUDA_A, float *CUDA_C) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
   //if (tid == 0){
     CUDA_C[tid] = CUDA_A[tid] + 3;
   //}
}

int simple_vector(int v){
  float A[3]= {2.0,4.0,6.0};
  float *C;
  float j = (float) v;

  C = (float*)malloc(sizeof(float) * N);
  A[0] =(float) v;
  printf("pasaste -> %f\n",j);
  float *CUDA_A, *CUDA_C;

  hipMalloc((void**)&CUDA_A, sizeof(float) * 4);
  hipMalloc((void**)&CUDA_C, sizeof(float) * 4);

  hipMemcpy(CUDA_A, A, sizeof(float)*4, hipMemcpyHostToDevice);

  simple_kernel<<<1,4>>>(CUDA_A, CUDA_C);

  hipMemcpy(C, CUDA_C, sizeof(float) * 4, hipMemcpyDeviceToHost); 

  printf("and_we_got-> %f\n",C[0]);
  printf("and_we_got-> %f\n",C[1]);
  printf("and_we_got-> %f\n",C[2]);
  printf("and_we_got-> %d\n",(int)(5.0*(C[0]+C[1]+C[2])));
  return (5.0)*(C[0]+C[1]+C[2]);
}

static VALUE get_number_from_card(VALUE self, VALUE value) {
  Check_Type(value, T_FIXNUM);

  int number_in = NUM2INT(value);
  int number_out = simple_vector(number_in);
  return INT2NUM(number_out);
}

extern "C"
void Init_nvidia()
{
  rb_define_global_function("print_length", rb_print_length, 1);
  rb_define_global_function("get_number_from_card", get_number_from_card, 1);
}

