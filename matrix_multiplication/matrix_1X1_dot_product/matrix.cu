#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>


#define N 512
#define MAX_ERR 1e-6

//__global__ void vector_add(float *out, float *a, float *b, int n) {
//   int stride = 1;	
//   int tid = blockIdx.x * blockDim.x + threadIdx.x;
           //  0        *    256     +    1 = 1  | BLOCK0 |  
           //  0        *    256     +    2 = 2 

           //  1        *    256     +    1 = 257 | BLOCK1 |   
           //  1        *    256     +    2 = 258

//   out[tid] = a[tid] + b[tid];   
//}

void print_results(float *C){
  printf("[");
  for(int i = 0 ; i < 4; i++){
    printf("%f,",C[i]);
  }
  printf("]\n");
}

__global__ void vector_add(float *CUDA_A, float *CUDA_B, float *CUDA_C, int n) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  CUDA_C[tid] = CUDA_A[tid] + CUDA_B[tid];   
}

__global__ void vector_sub(float *CUDA_A, float *CUDA_B, float *CUDA_C, int n) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  CUDA_C[tid] = CUDA_A[tid] - CUDA_B[tid];
}

__global__ void vector_dot_product(float *CUDA_A, float *CUDA_B, float *CUDA_C,float *CUDA_K, int n) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ float SHARED_K[1024];

  CUDA_C[tid] = CUDA_A[tid] * CUDA_B[tid];	
  // Only one kernel should apply the dot product
  __syncthreads();

  SHARED_K[tid] = CUDA_C[tid * 2] + CUDA_C[tid * 2 + 1];

  __syncthreads();
  if (tid == 0){
    *CUDA_K = SHARED_K[0] + SHARED_K[1]; 
  }
}

void print_vectors(){
  printf("A = {2.0,4.0,6.0,8.0}\n");
  printf("B = {1.0,2.0,3.0,10.0}\n");
};

int main(){
    float *C, *K;
    float *CUDA_A, *CUDA_B, *CUDA_C, *CUDA_K; 

    // Allocate host memory
    float A[4]= {2.0,4.0,6.0,10.0};
    float B[4]= {1.0,2.0,3.0,8.0};
    
    C = (float*)malloc(sizeof(float) * N);
    K = (float*)malloc(sizeof(float));

    // Allocate device memory
    hipMalloc((void**)&CUDA_A, sizeof(float) * N);
    hipMalloc((void**)&CUDA_B, sizeof(float) * N);
    hipMalloc((void**)&CUDA_C, sizeof(float) * N);
    hipMalloc((void**)&CUDA_C, sizeof(float) * N);
    hipMalloc((void**)&CUDA_K, sizeof(float));

    // Transfer data from host to device memory
    hipMemcpy(CUDA_A, A, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(CUDA_B, B, sizeof(float) * N, hipMemcpyHostToDevice);

    // Executing kernel 
    vector_add<<<1,4>>>(CUDA_A, CUDA_B, CUDA_C, N);
    hipMemcpy(C, CUDA_C, sizeof(float) * N, hipMemcpyDeviceToHost);

    //Executing kernel 
    puts("ADDING");
    print_vectors();
    print_results(C);

    vector_sub<<<1,4>>>(CUDA_A, CUDA_B, CUDA_C, N);
    hipMemcpy(C, CUDA_C, sizeof(float) * N, hipMemcpyDeviceToHost);
    puts("SUBSTRACTING");
    print_vectors();
    print_results(C);

    vector_dot_product<<<1,4>>>(CUDA_A, CUDA_B, CUDA_C, CUDA_K, N);
    hipMemcpy(C, CUDA_C, sizeof(float) * N, hipMemcpyDeviceToHost);
    puts("DOT_PRODUCT");
    print_vectors();

    print_results(C);

    hipMemcpy(K, CUDA_K, sizeof(float), hipMemcpyDeviceToHost);

    printf("\nDot product result %f\n", *K);
    // Deallocate device memory
    hipFree(CUDA_A);
    hipFree(CUDA_B);
    hipFree(CUDA_C);

    // Deallocate host memory
    //free(A); 
    //free(B); 
    free(C);
}

