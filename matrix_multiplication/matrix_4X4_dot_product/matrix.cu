#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define N 512
#define MAX_ERR 1e-6

//__global__ void vector_add(float *out, float *a, float *b, int n) {
//   int stride = 1;	
//   int tid = blockIdx.x * blockDim.x + threadIdx.x;
           //  0        *    256     +    1 = 1  | BLOCK0 |  
           //  0        *    256     +    2 = 2 

           //  1        *    256     +    1 = 257 | BLOCK1 |   
           //  1        *    256     +    2 = 258

//   out[tid] = a[tid] + b[tid];   
//}

void print_results(float *C){
  printf("[");
  for(int i = 0 ; i < 20; i++){
    printf("%f,",C[i]);
  }
  printf("]\n");
}

__global__ void vector_dot_product(float *CUDA_A, float *CUDA_B, float *CUDA_C,int n) {
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  __shared__ float SHARED_SUM[4];
  extern __shared__ float SHARED_PRODUCTS[4];
  int row = n;
  int col = n;
  int index = n;
  float *mul = (float *)malloc(sizeof(float) * n);
  float res;
  //Data is not visible to threads in other blocks you idiot!
  for(int i = 0; i < n; i++){
    //we need to flatten the stupid array 
    mul[i] = CUDA_A[bid * col + i] * CUDA_B[ i * row + tid];
  }
  //aggregate all the multiplications a11*b11 + .. + ann*bnn 
  for(int j = 0;j<n;j++){
    res += mul[j];
  }
  
  __syncthreads();
 
  CUDA_C[(bid*n) + tid] = res;
}

void print_vectors(){
  printf("A=[{2.0,5.0,2.0}\n");
  printf("   {1.0,2.0,4.0}]\n");
  printf("   {4.0,2.0,2.0}]\n");
  printf("B=[{3.0,5.0,3.0}\n");
  printf("   {2.0,3.0,5.0}]\n");
  printf("   {7.0,1.0,3.0}]\n");
};

int main(){
    float *C;
    float *CUDA_A, *CUDA_B,  *CUDA_C; 

    // Allocate host memory
    float A[4][4] = {
      {2,5,3,4},
      {1,2,3,2},
      {1,3,4,2},
      {1,3,4,4}
    };

    float B[4][4] = {
      {3,5,3,2},
      {2,3,2,3},
      {2,2,3,2},
      {2,2,2,3}
    };
    
    C = (float *)malloc(8*8*sizeof(float) * N);

    // Allocate device memory
    hipMalloc((void**)&CUDA_A, sizeof(float) * N);
    hipMalloc((void**)&CUDA_B, sizeof(float) * N);
    hipMalloc((void**)&CUDA_C, sizeof(float) * N);

    // Transfer data from host to device memory
    hipMemcpy(CUDA_A, A, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(CUDA_B, B, sizeof(float) * N, hipMemcpyHostToDevice);
    vector_dot_product<<<4,4>>>(CUDA_A, CUDA_B, CUDA_C, 4);
    hipMemcpy(C, CUDA_C, sizeof(float) * 500, hipMemcpyDeviceToHost);
    puts("DOT_PRODUCT");
    print_vectors();
    print_results(C);


    //printf("\nDot product result %f\n", *K);
    // Deallocate device memory
    hipFree(CUDA_A);
    hipFree(CUDA_B);
    hipFree(CUDA_C);

    // Deallocate host memory
    //free(A); 
    //free(B); 
    free(C);
}

