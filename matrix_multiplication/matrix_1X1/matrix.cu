#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>


#define N 512
#define MAX_ERR 1e-6

//__global__ void vector_add(float *out, float *a, float *b, int n) {
//   int stride = 1;	
//   int tid = blockIdx.x * blockDim.x + threadIdx.x;
           //  0        *    256     +    1 = 1  | BLOCK0 |  
           //  0        *    256     +    2 = 2 

           //  1        *    256     +    1 = 257 | BLOCK1 |   
           //  1        *    256     +    2 = 258

//   out[tid] = a[tid] + b[tid];   
//}

void print_results(float *C){
  printf("ADDING\n");
  printf("[");
  for(int i = 0 ; i < 3; i++){
    printf("%f,",C[i]);
  }
  printf("]\n");
}

void print_results_sub(float *C){
  printf("SUBSTRACTING\n");

   printf("[");
   for(int i = 0 ; i < 3; i++){
     printf("%f,",C[i]);
   }
   printf("]\n");
}

__global__ void vector_add(float *CUDA_A, float *CUDA_B, float *CUDA_C, int n) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  CUDA_C[tid] = CUDA_A[tid] + CUDA_B[tid];   
}

__global__ void vector_sub(float *CUDA_A, float *CUDA_B, float *CUDA_C, int n) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  CUDA_C[tid] = CUDA_A[tid] - CUDA_B[tid];
}

__global__ void vector_dot_product(float *CUDA_A, float *CUDA_B, float *CUDA_C,float *CUDA_K, int n) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  CUDA_C[tid] = CUDA_A[tid] * CUDA_B[tid];	
  // Only one kernel should apply the dot product
  __syncthreads();

  if(tid == 0){
    *CUDA_K = CUDA_C[tid] + CUDA_C[tid+1] + CUDA_C[tid+2]; 
  }
}

int main(){
    float *C, *K;
    float *CUDA_A, *CUDA_B, *CUDA_C, *CUDA_K; 

    // Allocate host memory
    float A[3]= {2.0,4.0,6.0};
    printf("A = {2.0,4.0,6.0}\n");
    float B[3]= {1.0,2.0,3.0};
    printf("B = {1.0,2.0,3.0}\n");

    C = (float*)malloc(sizeof(float) * N);
    K = (float*)malloc(sizeof(float));

    // Allocate device memory
    hipMalloc((void**)&CUDA_A, sizeof(float) * N);
    hipMalloc((void**)&CUDA_B, sizeof(float) * N);
    hipMalloc((void**)&CUDA_C, sizeof(float) * N);
    hipMalloc((void**)&CUDA_C, sizeof(float) * N);
    hipMalloc((void**)&CUDA_K, sizeof(float));

    // Transfer data from host to device memory
    hipMemcpy(CUDA_A, A, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(CUDA_B, B, sizeof(float) * N, hipMemcpyHostToDevice);

    // Executing kernel 
    vector_add<<<1,3>>>(CUDA_A, CUDA_B, CUDA_C, N);
    hipMemcpy(C, CUDA_C, sizeof(float) * N, hipMemcpyDeviceToHost);

    //Executing kernel 

    print_results(C);

    vector_sub<<<1,3>>>(CUDA_A, CUDA_B, CUDA_C, N);
    hipMemcpy(C, CUDA_C, sizeof(float) * N, hipMemcpyDeviceToHost);

    print_results_sub(C);

    vector_dot_product<<<1,3>>>(CUDA_A, CUDA_B, CUDA_C, CUDA_K, N);
    hipMemcpy(C, CUDA_C, sizeof(float) * N, hipMemcpyDeviceToHost);
    print_results(C);

    //cudaMemcpy(K, CUDA_K, sizeof(float), cudaMemcpyDeviceToHost);

    //printf("Dot product result %f", *K);
    // Deallocate device memory
    hipFree(CUDA_A);
    hipFree(CUDA_B);
    hipFree(CUDA_C);

    // Deallocate host memory
    //free(A); 
    //free(B); 
    free(C);
}

