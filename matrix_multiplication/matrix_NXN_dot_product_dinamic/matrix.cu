#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define MAX_ERR 1e-6

//__global__ void vector_add(float *out, float *a, float *b, int n) {
//   int stride = 1;	
//   int tid = blockIdx.x * blockDim.x + threadIdx.x;
           //  0        *    256     +    1 = 1  | BLOCK0 |  
           //  0        *    256     +    2 = 2 

           //  1        *    256     +    1 = 257 | BLOCK1 |   
           //  1        *    256     +    2 = 258

//   out[tid] = a[tid] + b[tid];   
//}

void print_results(float *ARRAY, int array_size){
  printf("[");
  for(int i = 0; i < array_size; i++){
    printf("{");	  
    for(int j = 0; j < array_size; j++){
      printf("%.1f,",ARRAY[(i * array_size) +j]); 
    }	    
    printf("}\n");	  
  }
  printf("]");
  printf("\n");
}

__global__ void vector_dot_product(float *CUDA_A, float *CUDA_B, float *CUDA_C,int n) {
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  __shared__ float SHARED_SUM[4];
  extern __shared__ float SHARED_PRODUCTS[4];
  int row = n;
  int col = n;
  int index = n;
  float *mul = (float *)malloc(sizeof(float) * n);
  float res;
  //Data is not visible to threads in other blocks you idiot!
  for(int i = 0; i < n; i++){
    //we need to flatten the stupid array 
    mul[i] = CUDA_A[bid * col + i] * CUDA_B[ i * row + tid];
  }
  //aggregate all the multiplications a11*b11 + .. + ann*bnn 
  for(int j = 0;j<n;j++){
    res += mul[j];
  }
  
  __syncthreads();
 
  CUDA_C[(bid*n) + tid] = res;
}

int main(){
    int array_size = 100;
    float *C, *A, *B;
    float *CUDA_A, *CUDA_B,  *CUDA_C; 

    
    A = (float *)malloc(array_size * array_size * sizeof(float));
    B = (float *)malloc(array_size * array_size * sizeof(float));
  
    float a = 5.0;

   for(int i=0;i<20000;i++){
    for(int i = 0; i<(array_size * array_size); i++){
      A[i] = ((float)rand()/(float)(RAND_MAX)) * a;
      B[i] = ((float)rand()/(float)(RAND_MAX)) * a;
    } 

    C = (float *)malloc(array_size * array_size *sizeof(float) );

    // Allocate device memory
    hipMalloc((void**)&CUDA_A, sizeof(float) * array_size * array_size);
    hipMalloc((void**)&CUDA_B, sizeof(float) * array_size * array_size);
    hipMalloc((void**)&CUDA_C, sizeof(float) * array_size * array_size);


       // Transfer data from host to device memory
      hipMemcpy(CUDA_A, A, sizeof(float) * array_size * array_size, hipMemcpyHostToDevice);
      hipMemcpy(CUDA_B, B, sizeof(float) * array_size * array_size, hipMemcpyHostToDevice);

      vector_dot_product<<<array_size,array_size>>>(CUDA_A, CUDA_B, CUDA_C, array_size);

      hipMemcpy(C, CUDA_C, sizeof(float) * array_size * array_size, hipMemcpyDeviceToHost);

      puts("DOT_PRODUCT");
      print_results(A,array_size);
      print_results(B,array_size);
      puts("C VALUE");
      print_results(C,array_size);

      // Deallocate device memory
      hipFree(CUDA_A);
      hipFree(CUDA_B);
      hipFree(CUDA_C);

      free(C);
    }

    
        // Deallocate host memory

}

