#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

int calculate_no_threads(int array_size){
 return 4;  
}

void print_results(double *ARRAY, int array_size){
  printf("[\n");
  for(int i = 0; i < array_size; i++){
    printf("{");	  
    for(int j = 0; j < array_size; j++){
      printf("%1.1f,",ARRAY[(i * array_size) +j]); 
    }	    
    printf("}\n");	  
  }
  printf("]");
  printf("\n");
}

__global__ void vector_dot_product(double *CUDA_A, double *CUDA_B, double *CUDA_C,double *CUDA_T,int array_size,int no_threads) {
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int row_no = array_size;
  int col_no = array_size;
  float todo=0;
  //Make multiplications 
  for (int p = 0; p < array_size; p++){
    for(int i = 0; i < array_size; i++){
      for (int j = 0; j < array_size; j++){	  
	todo = todo + CUDA_A[p * col_no + j] * CUDA_B[ j * row_no + i];
      }	    

      CUDA_T[chin++]=todo;
      todo = 0;
    }
  }
}

int main(){
    //int array_size = 7900;
    int array_size = 4;
    double *C, *A, *B, *T;
    double *CUDA_A, *CUDA_B,  *CUDA_C, *CUDA_T; 
    
    A = (double *)malloc(array_size * array_size * sizeof(double));
    B = (double *)malloc(array_size * array_size * sizeof(double));
    T = (double *)malloc((array_size*array_size) * sizeof(double));
    C = (double *)malloc(array_size * array_size *  sizeof(double) );
  
    double a = 1.5;

    for(int i = 0; i<(array_size * array_size); i++){
      A[i] = ((double)rand()/(double)(RAND_MAX)) * a;
      B[i] = ((double)rand()/(double)(RAND_MAX)) * a;
    } 

    // Allocate device memory
    hipMalloc((void**)&CUDA_A, sizeof(double) * array_size * array_size);
    hipMalloc((void**)&CUDA_B, sizeof(double) * array_size * array_size);
    hipMalloc((void**)&CUDA_C, sizeof(double) * array_size * array_size);
    hipMalloc((void**)&CUDA_T, sizeof(double) * (array_size*array_size));

    // Transfer data from host to device memory
    hipMemcpy(CUDA_A, A, sizeof(double) * array_size * array_size, hipMemcpyHostToDevice);
    hipMemcpy(CUDA_B, B, sizeof(double) * array_size * array_size, hipMemcpyHostToDevice);
    hipMemcpy(CUDA_T, T, sizeof(double) * array_size * array_size, hipMemcpyHostToDevice);

    printf("calculate_no_threads %d\n",calculate_no_threads(array_size)); 
    vector_dot_product<<<1,calculate_no_threads(array_size)>>>(CUDA_A, CUDA_B, CUDA_C, CUDA_T,array_size,calculate_no_threads(array_size));

    hipMemcpy(C, CUDA_C, sizeof(double) * array_size * array_size, hipMemcpyDeviceToHost);
    hipMemcpy(T, CUDA_T, sizeof(double) * (array_size*array_size), hipMemcpyDeviceToHost);

    puts("DOT_PRODUCT");
    print_results(A,array_size);
    print_results(B,array_size);

    puts("MATRIX MULTI");
    print_results(T,array_size);
    //for(int i =0;i<array_size*array_size;i++){
    //  printf("%f,",T[i]);
   // }


    // Deallocate device memory
    hipFree(CUDA_A);
    hipFree(CUDA_B);
    hipFree(CUDA_C);
    hipFree(CUDA_T);

    free(C);
    free(A);
    free(B);
    free(T);
    // Deallocate host memory
}

