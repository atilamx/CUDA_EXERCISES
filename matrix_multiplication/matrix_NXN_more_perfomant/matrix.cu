#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>


int calculate_no_threads(int array_size){
 if(array_size<256){
   return array_size;	 
 } else {
   return 1024;
 }
}

void print_results(double *ARRAY, int array_size){
  printf("[\n");
  for(int i = 0; i < array_size; i++){
    printf("{");	  
    for(int j = 0; j < array_size; j++){
      printf("%1.1lf,",ARRAY[(i * array_size) +j]); 
    }	    
    printf("}\n");	  
  }
  printf("]");
  printf("\n");
}

__global__ void vector_dot_product(double *CUDA_A, double *CUDA_B, double *CUDA_C,double *CUDA_T,int array_size,int no_threads) {
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  
  int row_count = array_size;
  int col_count = array_size;
  int NumberThreads = no_threads;
  int batch = array_size/NumberThreads;
  int Remaninder = array_size%NumberThreads;
  
  int StartRow;
  int EndRow; 
  
  StartRow = batch * tid; //For testing replace tid with 0..n batch

  if (StartRow == 0){
    EndRow = StartRow + batch + Remaninder;
  } else {
    StartRow = StartRow + Remaninder;
    EndRow = StartRow + batch;
  }

  int StarTingPoint =  array_size*StartRow;

  int increment = 0;
  float product = 0;

  for (int row = StartRow; row < EndRow; row++){
    for(int column = 0; column < array_size; column++){
      for (int cell = 0; cell < array_size; cell++){	  
	product = product + CUDA_A[row * col_count + cell] * CUDA_B[ cell * row_count + column];
      }	    

      CUDA_T[(StarTingPoint)+increment++] = product;
      product = 0;
    }
  }
   __syncthreads();
}

int main(){
    //int array_size = 7900;
    int array_size = 3000;
    double *C, *A, *B, *T;
    double *CUDA_A, *CUDA_B,  *CUDA_C, *CUDA_T; 
    
    A = (double *)malloc(array_size * array_size * sizeof(double));
    B = (double *)malloc(array_size * array_size * sizeof(double));
    T = (double *)malloc((array_size*array_size) * sizeof(double));
    C = (double *)malloc(array_size * array_size *  sizeof(double) );
  
    double a = 0.5;

    for(int i = 0; i<(array_size * array_size); i++){
      A[i] = ((double)rand()/(double)(RAND_MAX)) * a;
      B[i] = ((double)rand()/(double)(RAND_MAX)) * a;
    } 

    // Allocate device memory
    hipMalloc((void**)&CUDA_A, sizeof(double) * array_size * array_size);
    hipMalloc((void**)&CUDA_B, sizeof(double) * array_size * array_size);
    hipMalloc((void**)&CUDA_C, sizeof(double) * array_size * array_size);
    hipMalloc((void**)&CUDA_T, sizeof(double) * (array_size*array_size));

    // Transfer data from host to device memory
    hipMemcpy(CUDA_A, A, sizeof(double) * array_size * array_size, hipMemcpyHostToDevice);
    hipMemcpy(CUDA_B, B, sizeof(double) * array_size * array_size, hipMemcpyHostToDevice);
    hipMemcpy(CUDA_T, T, sizeof(double) * array_size * array_size, hipMemcpyHostToDevice);

    printf("calculate_no_threads %d\n",calculate_no_threads(array_size)); 
    vector_dot_product<<<1,calculate_no_threads(array_size)>>>(CUDA_A, CUDA_B, CUDA_C, CUDA_T,array_size,calculate_no_threads(array_size));

    hipMemcpy(C, CUDA_C, sizeof(double) * array_size * array_size, hipMemcpyDeviceToHost);
    hipMemcpy(T, CUDA_T, sizeof(double) * (array_size*array_size), hipMemcpyDeviceToHost);

    puts("DOT_PRODUCT");
    print_results(A,array_size);
    print_results(B,array_size);

    puts("MATRIX MULTI");
    print_results(T,array_size);


    // Deallocate device memory
    hipFree(CUDA_A);
    hipFree(CUDA_B);
    hipFree(CUDA_C);
    hipFree(CUDA_T);

    free(C);
    free(A);
    free(B);
    free(T);
    // Deallocate host memory
}

