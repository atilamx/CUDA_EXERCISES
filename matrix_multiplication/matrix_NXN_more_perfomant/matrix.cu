#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>


#define MAX_ERR 1e-6

//__global__ void vector_add(float *out, float *a, float *b, int n) {
//   int stride = 1;	
//   int tid = blockIdx.x * blockDim.x + threadIdx.x;
           //  0        *    256     +    1 = 1  | BLOCK0 |  
           //  0        *    256     +    2 = 2 

           //  1        *    256     +    1 = 257 | BLOCK1 |   
           //  1        *    256     +    2 = 258

//   out[tid] = a[tid] + b[tid];   
//}
int calculate_no_threads(int array_size){
  if(array_size/256 < 1){
    return 1;  
  } else {
    return array_size/256;
  }
}

void print_results(float *ARRAY, int array_size){
  printf("[");
  for(int i = 0; i < array_size; i++){
    printf("{");	  
    for(int j = 0; j < array_size; j++){
      printf("%1.1f,",ARRAY[(i * array_size) +j]); 
    }	    
    printf("}\n");	  
  }
  printf("]");
  printf("\n");
}

__global__ void vector_dot_product(float *CUDA_A, float *CUDA_B, float *CUDA_C,float *CUDA_T,int array_size,int no_threads) {
  int tid = threadIdx.x;
  int bid = blockIdx.x;

  int row_no = array_size;
  int col_no = array_size;
  float *mul = (float *)malloc(sizeof(float) * array_size);
  double *sum = (double *)malloc(sizeof(double) * 300);

  //Make multiplications 
  for (int p = 0; p < array_size; p++){
    for(int i = 0; i < array_size; i++){
      for (int j = 0; j < array_size; j++){	  
        mul[((i*array_size)+j) + p*array_size*array_size] = CUDA_A[p * col_no + j] * CUDA_B[ j * row_no + i];
      }	    
    }
  }
   
  float res=0.0;
   //sum all multiplications a1.a2+b1.b3
   for(int r=0;r<array_size;r++){
    for(int j=0;j<array_size;j++){
      for(int k=0;k<array_size;k++){
        res += mul[k+(j*array_size)+(r*(array_size*array_size))]; 	  
      }

      sum[j+(r*array_size)] = res;
      res = 0;
    }
   }

  for(int j = 0;j<300;j++){
    //CUDA_C[(i*array_size) + j] = mul[(i*row_no)+j]; 
    //place all the results back to the array	  
    CUDA_T[j] = sum[j]; 
  }
  
  __syncthreads();
 
}

int main(){
    int array_size = 3;
    float *C, *A, *B, *T;
    float *CUDA_A, *CUDA_B,  *CUDA_C, *CUDA_T; 
    
    A = (float *)malloc(array_size * array_size * sizeof(float));
    B = (float *)malloc(array_size * array_size * sizeof(float));
    T = (float *)malloc(300 * sizeof(float));
  
    float a = 4.0;

    for(int i = 0; i<(array_size * array_size); i++){
      A[i] = ((float)rand()/(float)(RAND_MAX)) * a;
      B[i] = ((float)rand()/(float)(RAND_MAX)) * a;
    } 

    //Fill remaining bytes in array with 1s
    //for(int i = 0; i<300;i++){
    //  T[i] = 1;
   // }

    C = (float *)malloc(array_size * array_size *  sizeof(float) );

    // Allocate device memory
    hipMalloc((void**)&CUDA_A, sizeof(float) * array_size * array_size);
    hipMalloc((void**)&CUDA_B, sizeof(float) * array_size * array_size);
    hipMalloc((void**)&CUDA_C, sizeof(float) * array_size * array_size);
    hipMalloc((void**)&CUDA_T, sizeof(float) * 300);

    // Transfer data from host to device memory
    hipMemcpy(CUDA_A, A, sizeof(float) * array_size * array_size, hipMemcpyHostToDevice);
    hipMemcpy(CUDA_B, B, sizeof(float) * array_size * array_size, hipMemcpyHostToDevice);
    hipMemcpy(CUDA_T, T, sizeof(float) * array_size * array_size, hipMemcpyHostToDevice);

    printf("calculate_no_threads %d\n",calculate_no_threads(array_size)); 
    vector_dot_product<<<1,calculate_no_threads(array_size)>>>(CUDA_A, CUDA_B, CUDA_C, CUDA_T,array_size,calculate_no_threads(array_size));

    hipMemcpy(C, CUDA_C, sizeof(float) * array_size * array_size, hipMemcpyDeviceToHost);
    hipMemcpy(T, CUDA_T, sizeof(float) * 300, hipMemcpyDeviceToHost);

    puts("DOT_PRODUCT");
    print_results(A,array_size);
    print_results(B,array_size);

    puts("MATRIX MULTI");
    print_results(T,array_size);

    // Deallocate device memory
    hipFree(CUDA_A);
    hipFree(CUDA_B);
    hipFree(CUDA_C);

    free(C);
    // Deallocate host memory
}

